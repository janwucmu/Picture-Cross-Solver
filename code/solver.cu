#include "hip/hip_runtime.h"
#include "solver_seq.h"
#include <algorithm>
#include <string>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

int old_dim_x;
int old_dim_y;

static int _argc;
static const char **_argv;
#define thread_size 32

__global__ const char *get_option_string(const char *option_name, const char *default_value) {
    for (int i = _argc - 2; i >= 0; i -= 2)
        if (strcmp(_argv[i], option_name) == 0)
            return _argv[i + 1];
    return default_value;
}

int get_option_int(const char *option_name, int default_value) {
    for (int i = _argc - 2; i >= 0; i -= 2)
        if (strcmp(_argv[i], option_name) == 0)
            return atoi(_argv[i + 1]);
    return default_value;
}

__global__ float get_option_float(const char *option_name, float default_value) {
    for (int i = _argc - 2; i >= 0; i -= 2)
        if (strcmp(_argv[i], option_name) == 0)
            return (float)atof(_argv[i + 1]);
    return default_value;
}

__global__ void  print_hints(thrust::device_vector<thrust::device_vector<int>> hints) {
    for (int j = 0; j < hints.size(); j++) {
        for (int k = 0; k < hints[j].size(); k++) {
            printf("%d ", hints[j][k]);
        }
        printf("\n");
    }    
}
void print_puzzle(pic_cross_t pic_cross) {  
    int* puzzle = pic_cross.puzzle;
    int dim_x = pic_cross.dim_x;
    int dim_y = pic_cross.dim_y;
    for (int i = 0; i < dim_x; i++) {
        std::bitset<64> x(puzzle[i]);
            for (int k = dim_y - 1; k != -1; k --) {
                std::cout << x[k];
            }
            std::cout << "\n";
        }
}
__global__ void  print_2d(int** col) { 
    for (int i = 0; i < old_dim_x; i++) {
        for (int j = 0; j < old_dim_y; j++) {
            std::cout << col[i][j];
        }
        std::cout << "\n";
    }
}
__global__ void  print_row_perm(thrust::device_vector<thrust::device_vector<int>> Row_perm, int dim_y) {
    for (int i = 0; i < Row_perm.size(); i ++) {
        for (int j = 0; j < Row_perm[i].size(); j ++) {
            std::bitset<64> x(Row_perm[i][j]);
            for (int k = dim_y - 1; k != -1; k --) {
                std::cout << x[k];
            }
            std::cout << " ";
        }
            std::cout << "\n";
        }
}
__global__ void  write_output(int argc, const char *argv[], pic_cross_t pic_cross) {
    _argc = argc - 1;
    _argv = argv + 1;
    const char *input_filename = get_option_string("-f", NULL);
    int* puzzle = pic_cross.puzzle;
    int dim_x = pic_cross.dim_x;
    int dim_y = pic_cross.dim_y;

    std::string filename_long = ((std::string) input_filename);
    int index =filename_long.find_last_of("/");
    std::string filename = filename_long.substr(index + 1, (filename_long.size() - 4 - index - 1));
    std::ofstream output1;
    output1.open((std::string)"outputs/output_" + filename + ".txt");
    output1 << dim_x << " " << dim_y << "\n";
    for (int i = 0; i < dim_x; i++) {
        std::bitset<64> x(puzzle[i]);
            for (int k = dim_y - 1; k != -1; k --) {
                output1 << x[k];
            }
            output1 << "\n";
        }
    output1.close();
}
__global__ pic_cross_t read_input(int argc, const char *argv[]) {
    int dim_x, dim_y;
    pic_cross_t pic_cross;
    int temp = 1;
    
    _argc = argc - 1;
    _argv = argv + 1;
    const char *input_filename = get_option_string("-f", NULL);
    FILE *input = fopen(input_filename, "r");
    if (!input) {
        printf("Unable to open file: %s.\n", input_filename);
        return pic_cross;
    }
    fscanf(input, "%d %d\n", &dim_x, &dim_y);
    pic_cross.puzzle = (int*)calloc(dim_x, sizeof(int));
    pic_cross.dim_x = dim_x;
    pic_cross.dim_y = dim_y;
    thrust::device_vector<thrust::device_vector<int>> hints;
    hints.resize(dim_x + dim_y);
    std::ifstream file(input_filename);
    std::string line;
    int i = 0;
    while(getline(file, line)) {
        std::istringstream ss(line);
        int num;
        if (temp == 1) {
            temp = 0;
            continue;
        }
        while (ss >> num) {
            hints[i].push_back(num);
        }
        i++;
    }
    pic_cross.hints = hints;
    return pic_cross;
}

// void updatedim_xols(int row, int* grid, int* colVal, int* colIdx, int* cols){
//     float ixc = 1;
//     for(int c = 0; c < dim_x; c++){
//         // copy from previous
//         colVal[(row * dim_x) + c] = row==0 ? 0 : colVal[(row-1) * dim_x + c];
//         colIdx[row * dim_x + c] = row==0 ? 0 : colIdx[(row-1) * dim_x + c];
//         if((grid[row] & ixc)==0){
//             if(row > 0 && colVal[(row-1) * dim_x + c] > 0){ 
//                 // bit not set and col is not empty at previous row => close blocksize
//                 colVal[row * dim_x + c]=0;
//                 colIdx[row * dim_x + c]++;
//             }
//         }
//         else{
//             colVal[row * dim_x + c]++; // increase value for set bit
//         }
//         ixc <<= 1;
//     }
// }


__global__ float bits(int b){
    return (1 << b) - 1; // 1 => 1, 2 => 11, 3 => 111, ...
}

__global__ void  calcPerms(int r, int cur, int spaces, std::size_t perm, int shift, pic_cross_t pic_cross, thrust::device_vector<int> &res){
    // int dim_x = pic_cross.dim_x;
    // int dim_y = pic_cross.dim_y;
    // int* puzzle = pic_cross.puzzle;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_index = threadIdx.y * blockDim.x + threadIdx.x; 

    thrust::device_vector<thrust::device_vector<int>> hints = pic_cross.hints;

    if(cur == hints[r].size()){
        // if((puzzle[r] & perm) == puzzle[r]){
        //     res.add(perm);				
        // }
        res.push_back(perm);
        return;
    }
    while(spaces >= 0){
        int b = bits(hints[r][cur]);
        calcPerms(r, cur+1, spaces, perm|(b<<shift), shift+hints[r][cur]+1, pic_cross, res);
        shift++;
        spaces--;
    }
}

// at every row and column (every box in the grid)
// colVal[r][c]: current position within the current blocksize
// colIx[r][c]: current block index
// The value increased by 1 if the column is painted in the current row.
// The value reset to 0 and index increased by 1 if the column was painted in the previous row and is not in the current row.
__global__ void  updateCols(int row, int numCol, int* puzzle, int** colVal, int** colIx) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_index = threadIdx.y * blockDim.x + threadIdx.x;
    int ixc = (int)pow(2, numCol-1);
    for (int c = 0; c < numCol; c++) {

        colVal[row][c] = (row == 0) ? 0 : colVal[row-1][c];
        colIx[row][c] = (row == 0) ? 0 : colIx[row-1][c];
        if ((puzzle[row] & ixc) == 0) {

            if ((row > 0) && (colVal[row-1][c] > 0)) {

                colVal[row][c] = 0;
                colIx[row][c]++; 
            }
        }
        else {
            
            colVal[row][c]++;
        }
        ixc >>= 1;
    }
}

__global__ void  rowMask(int row, int numCol, long* mask, long* val, 
             int** colVal,
             int** colIx, pic_cross_t* pic_cross) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_index = threadIdx.y * blockDim.x + threadIdx.x;
    int dim_x = pic_cross->dim_x;
    int dim_y = pic_cross->dim_y;
    thrust::device_vector<thrust::device_vector<int>> hints = pic_cross->hints;
    mask[row] = 0;
    val[row] = 0;
    if (row == 0) {
        return;
    }
    int ixc = (int)pow(2, numCol-1);
    for (int c = 0; c < dim_y; c++) {
        if (colVal[row-1][c] > 0) {
            mask[row] |= ixc;
            int index = colIx[row-1][c];
            // printf("c: %d row: %d hints: %d colVal: %d \n", c, row, hints[c+dim_x][index], colVal[row-1][c]);
            if (hints[c + dim_x][index] > colVal[row-1][c]) {
                val[row] |= ixc;
            }
        }
        else if (colVal[row-1][c] == 0 && colIx[row-1][c] == hints[c+dim_x].size()) {
            mask[row] |= ixc;
        }
        ixc >>= 1;
    }
}



__global__ bool dfs(int row, thrust::device_vector<thrust::device_vector<int>>& Row_perm, 
        long* mask,
        long* val,
        int** colVal,
        int** colIx,
        pic_cross_t * pic_cross){
    int dim_x = pic_cross->dim_x;
    int dim_y = pic_cross->dim_y;
    int* puzzle = pic_cross->puzzle;
    int check = std::rand() %  Row_perm[row].size();
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_index = threadIdx.y * blockDim.x + threadIdx.x;

    if (row == dim_x) {
        return true;
    }

    rowMask(row, dim_y, mask, val, colVal, colIx, pic_cross);

    for (int i = 0; i < Row_perm[row].size();i++) {
        if ((Row_perm[row][i] & mask[row]) != val[row] && i != check) {
            continue;
        }
        __syncthreads();
        puzzle[row] = Row_perm[row][i];

        updateCols(row, dim_y, puzzle, colVal, colIx);
        if (dfs(row+1, Row_perm, mask, val, colVal, colIx, pic_cross)) {   
            return true;
        }
    }
    return false;
}

int main(int argc, const char *argv[]) {

   const int bytes = sizeof(float) * N;
    hipMalloc(&device_x, bytes);  // allocate array in device memory
    hipMalloc(&device_y, bytes);      // allocate array in device memory
    hipMalloc(&device_result, bytes);      // allocate array in device memory
    double startTime = CycleTimer::currentSeconds();


    auto init_start = Clock::now();
    double t_time = 0;
    pic_cross_t pic_cross = read_input(argc, argv);
    if (pic_cross.dim_x == 0) {
        return 1;
    }
    // print_hints(pic_cross.hints);
    int dim_x = pic_cross.dim_x;
    int dim_y = pic_cross.dim_y;
    old_dim_x = dim_x;
    old_dim_y = dim_y;
    thrust::device_vector<thrust::device_vector<int>> hints = pic_cross.hints;
    thrust::device_vector<thrust::device_vector<int>> Row_perm;
    //Precal stuff
    thrust::device_vector<int> res;
    for (int r = 0; r < dim_x; r++) {
        res.clear();
        int space = dim_y - (hints[r].size() - 1);
        for (int i = 0; i < hints[r].size(); i ++) {
            space -= hints[r][i];
        }
        calcPerms(r, 0, space, 0, 0, pic_cross, res);
        Row_perm.push_back(res);
    }
    hipDeviceSynchronize();
    hipMemcpy(resultarray, device_result, bytes, hipMemcpyDeviceToHost);


    int** colVal = (int**) calloc(dim_x , sizeof(int*)) ;
    int** colIx = (int**) calloc(dim_x, sizeof(int*));
    for (int i = 0; i < dim_y; i++) {
        colVal[i] = (int*) calloc(dim_y, sizeof(int));
        colIx[i] = (int*) calloc(dim_y, sizeof(int));
    }
    long* mask = (long*) calloc(dim_x, sizeof(long));
    long* val = (long*) calloc(dim_x, sizeof(long));

    if (dfs(0, Row_perm, mask, val, colVal, colIx, &pic_cross)) {
    };
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    hipError_t errCode = hipPeekAtLastError();

    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    write_output(argc, argv, pic_cross);
    
}
void printCudaInfo() {

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}